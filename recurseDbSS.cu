
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 128

__global__ void naiveSS(int *in, int *out) {
    __shared__ int source[SIZE];
    __shared__ int destination[SIZE];
    __shared__ int temp[SIZE];

    int tIdx = threadIdx.x;
    source[tIdx] = in[tIdx];

    for (int i = 1; i < SIZE; i *= 2){
        __syncthreads();
        if (tIdx < i){
            destination[tIdx] = source[tIdx];
        }
        else{
            destination[tIdx] = source[tIdx] + source[tIdx - (i)];

        }
        __syncthreads();
        temp[tIdx] = destination[tIdx];
        destination[tIdx] = source[tIdx];
        source[tIdx] = temp[tIdx];
    }

    out[tIdx] = source[tIdx];

}

int main() {
    // timer information source: https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
    // timer allocation
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);


    // allocate memory
    int *input, *output;
    hipMallocManaged(&input, sizeof(int) * SIZE);
    hipMallocManaged(&output, sizeof(int) * SIZE);

    // initialize inputs
    for (int i = 0; i < SIZE; i++) {
        input[i] = 1;
    }
    // // test non ones input
    // input[0] = 3;
    // input[1] = 1;
    // input[2] = 7;
    // input[3] = 0;
    // input[4] = 4;
    // input[5] = 1;
    // input[6] = 6;
    // input[7] = 3;

    // collect first timer dp
    hipEventRecord(start);
    // run the kernel
    naiveSS<<<1, SIZE>>>(input, output);
    hipDeviceSynchronize();
    // collect second timer dp
    hipEventRecord(end);

    hipEventSynchronize(end);
    float milsec = 0;
    // calculates total run time in ms
    hipEventElapsedTime(&milsec, start, end);

    printf("elapsed time: %f ms\n", milsec);

    // // check results
    // for (int i = 0; i < SIZE; i++) {
    //     printf("%d ", output[i]);
    // }
    // printf("\n");

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // free mem
    hipFree(input);
    hipFree(output);

    return 0;
}