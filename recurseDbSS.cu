
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 8

__global__ void naiveSS(int *in, int *out) {
    __shared__ int source[SIZE];
    __shared__ int destination[SIZE];
    __shared__ int temp[SIZE];

    int tIdx = threadIdx.x;
    source[tIdx] = in[tIdx];

    for (int i = 1; i < SIZE; i *= 2){
        __syncthreads();
        if (tIdx < i){
            destination[tIdx] = source[tIdx];
        }
        else{
            destination[tIdx] = source[tIdx] + source[tIdx - (i)];

        }
        __syncthreads();
        temp[tIdx] = destination[tIdx];
        destination[tIdx] = source[tIdx];
        source[tIdx] = temp[tIdx];
    }

    out[tIdx] = source[tIdx];

}

int main() {
    // allocate memory
    int *input, *output;
    hipMallocManaged(&input, sizeof(int) * SIZE);
    hipMallocManaged(&output, sizeof(int) * SIZE);

    // // initialize inputs
    // for (int i = 0; i < SIZE; i++) {
    //     input[i] = 1;
    // }
    // input[0] = 3;
    // input[1] = 1;
    // input[2] = 7;
    // input[3] = 0;
    // input[4] = 4;
    // input[5] = 1;
    // input[6] = 6;
    // input[7] = 3;

    naiveSS<<<1, SIZE>>>(input, output);
    hipDeviceSynchronize();

    // check results
    for (int i = 0; i < SIZE; i++) {
        printf("%d ", output[i]);
    }
    printf("\n");

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // free mem
    hipFree(input);
    hipFree(output);

    return 0;
}