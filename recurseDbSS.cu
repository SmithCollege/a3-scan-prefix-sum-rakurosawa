
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define SIZE 512 // test 128, 256, 512

__global__ void naiveSS(int *in, int *out) {
    __shared__ int source[SIZE];
    __shared__ int destination[SIZE];
    __shared__ int temp[SIZE];

    int tIdx = threadIdx.x;
    source[tIdx] = in[tIdx];

    for (int i = 1; i < SIZE; i *= 2){
        __syncthreads();
        if (tIdx < i){
            destination[tIdx] = source[tIdx];
        }
        else{
            destination[tIdx] = source[tIdx] + source[tIdx - (i)];
        }
        __syncthreads();
        temp[tIdx] = destination[tIdx];
        destination[tIdx] = source[tIdx];
        source[tIdx] = temp[tIdx];
    }

    out[tIdx] = source[tIdx];

}

double get_clock() {
    struct timeval tv; int ok;
    ok = gettimeofday(&tv, (void *) 0);
    if (ok<0) { 
        printf("gettimeofday error"); 
        }
    return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main() {


    // allocate memory
    int *input, *output;
    hipMallocManaged(&input, sizeof(int) * SIZE);
    hipMallocManaged(&output, sizeof(int) * SIZE);

    double t0, t1;

    // initialize inputs
    for (int i = 0; i < SIZE; i++) {
        input[i] = 1;
    }
    // // test non ones input
    // input[0] = 3;
    // input[1] = 1;
    // input[2] = 7;
    // input[3] = 0;
    // input[4] = 4;
    // input[5] = 1;
    // input[6] = 6;
    // input[7] = 3;

    // collect first timer dp
    t0 = get_clock();
    // run the kernel
    naiveSS<<<1, SIZE>>>(input, output);
    hipDeviceSynchronize();
    // collect second timer dp
    t1 = get_clock();

    printf("time: %f ns\n", 1000000000.0*(t1-t0));

    // // check results
    // for (int i = 0; i < SIZE; i++) {
    //     printf("%d ", output[i]);
    // }
    // printf("\n");

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // free mem
    hipFree(input);
    hipFree(output);

    return 0;
}