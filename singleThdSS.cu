// include any headers

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define SIZE 128

__host__ void singleThdSS(int *in, int *out) {
    // copy the first value of in to out
    out[0] = in[0];
    // add the prior value in out to the next value in in 
    for (int i = 1; i < SIZE; i++){
        out[i] = out[i-1] + in[i];
    }
}

int main(void){
    // allocate memory
    int *input, *output;
    hipMallocManaged(&input, SIZE*sizeof(int));
    hipMallocManaged(&output, SIZE*sizeof(int));

    // initialize inputs
    for (int i = 0; i < SIZE; i++) {
        input[i] = 1;
    }

    singleThdSS(input, output);

    // // check results
    // for (int i = 0; i < SIZE; i++) {
    //     printf("%d ", output[i]);
    // }
    // printf("\n");

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // free memory
    hipFree(input);
    hipFree(output);

    return 0;
}