
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>

#define SIZE 128 // test 128, 256, 512

__host__ void singleThdSS(int *in, int *out) {
    // copy the first value of in to out
    out[0] = in[0];
    // add the prior value in out to the next value in in 
    for (int i = 1; i < SIZE; i++){
        out[i] = out[i-1] + in[i];
    }
}


double get_clock() {
    struct timeval tv; int ok;
    ok = gettimeofday(&tv, (void *) 0);
    if (ok<0) { 
        printf("gettimeofday error"); 
        }
    return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}


int main(void){

    // allocate memory
    int *input, *output;
    hipMallocManaged(&input, SIZE*sizeof(int));
    hipMallocManaged(&output, SIZE*sizeof(int));

    double t0, t1;

    // initialize inputs
    for (int i = 0; i < SIZE; i++) {
        input[i] = 1;
    }

    // collect first timer dp
    t0 = get_clock();
    // run the program
    singleThdSS(input, output);
    // collect second timer dp
    t1 = get_clock();

    printf("time: %f ns\n", 1000000000.0*(t1-t0));

    // // check results
    // for (int i = 0; i < SIZE; i++) {
    //     printf("%d ", output[i]);
    // }
    // printf("\n");

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // free memory
    hipFree(input);
    hipFree(output);

    return 0;
}