// include any headers

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define SIZE 128

__host__ void singleThdSS(int *in, int *out) {
    // copy the first value of in to out
    out[0] = in[0];
    // add the prior value in out to the next value in in 
    for (int i = 1; i < SIZE; i++){
        out[i] = out[i-1] + in[i];
    }
}

int main(void){
    // timer information source: https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
    // timer allocation
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // allocate memory
    int *input, *output;
    hipMallocManaged(&input, SIZE*sizeof(int));
    hipMallocManaged(&output, SIZE*sizeof(int));

    // initialize inputs
    for (int i = 0; i < SIZE; i++) {
        input[i] = 1;
    }

    // collect first timer dp
    hipEventRecord(start);
    // run the program
    singleThdSS(input, output);
    // collect second timer dp
    hipEventRecord(end);

    hipEventSynchronize(end);
    float milsec = 0;
    // calculates total run time in ms
    hipEventElapsedTime(&milsec, start, end);

    printf("elapsed time: %f ms\n", milsec);

    // // check results
    // for (int i = 0; i < SIZE; i++) {
    //     printf("%d ", output[i]);
    // }
    // printf("\n");

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // free memory
    hipFree(input);
    hipFree(output);

    return 0;
}