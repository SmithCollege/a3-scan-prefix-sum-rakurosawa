
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 128

__global__ void naiveSS(int *in, int *out) {
    for (int i = 0; i < SIZE; i++) {
        int value = 0;
        for (int j = 0; j <= i; j++) {
            value += in[j];
        }
        out[i] = value;
    }

}

int main() {
    // allocate memory
    int *input, *output;
    hipMallocManaged(&input, sizeof(int) * SIZE);
    hipMallocManaged(&output, sizeof(int) * SIZE);

    // initialize inputs
    for (int i = 0; i < SIZE; i++) {
        input[i] = 1;
    }

    naiveSS<<<1, 1>>>(input, output);
    hipDeviceSynchronize();

    // // check results
    // for (int i = 0; i < SIZE; i++) {
    //     printf("%d ", output[i]);
    // }
    // printf("\n");

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // free mem
    hipFree(input);
    hipFree(output);

    return 0;
}