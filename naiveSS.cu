
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define SIZE 128 // test 128, 256, 512

__global__ void naiveSS(int *in, int *out) { // change to float for scaling test
    for (int i = 0; i < SIZE; i++) {
        int value = 0;
        for (int j = 0; j <= i; j++) {
            value += in[j];
        }
        out[i] = value;
    }

}


double get_clock() {
    struct timeval tv; int ok;
    ok = gettimeofday(&tv, (void *) 0);
    if (ok<0) { 
        printf("gettimeofday error"); 
    }
    return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main() {

    // allocate memory
    int *input, *output; // change to float for scaling test
    hipMallocManaged(&input, sizeof(int) * SIZE); // change to float for scaling test
    hipMallocManaged(&output, sizeof(int) * SIZE); // change to float for scaling test

    double t0, t1;

    // initialize inputs
    for (int i = 0; i < SIZE; i++) {
        input[i] = 1; // change to 1.0 for scaling test
    }

    // collect first timer dp
    t0 = get_clock();
    // run the kernel
    naiveSS<<<1, 1>>>(input, output);
    hipDeviceSynchronize();
    // collect second timer dp
    t1 = get_clock();

   printf("time: %f ns\n", 1000000000.0*(t1-t0));

    // // check results
    // for (int i = 0; i < SIZE; i++) {
    //     printf("%d ", output[i]);
    // }
    // printf("\n");

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // free mem
    hipFree(input);
    hipFree(output);

    return 0;
}