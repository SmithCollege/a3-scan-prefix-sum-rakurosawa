
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 128

__global__ void naiveSS(int *in, int *out) {
    for (int i = 0; i < SIZE; i++) {
        int value = 0;
        for (int j = 0; j <= i; j++) {
            value += in[j];
        }
        out[i] = value;
    }

}

int main() {
    // timer information source: https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
    // timer allocation
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // allocate memory
    int *input, *output;
    hipMallocManaged(&input, sizeof(int) * SIZE);
    hipMallocManaged(&output, sizeof(int) * SIZE);

    // initialize inputs
    for (int i = 0; i < SIZE; i++) {
        input[i] = 1;
    }

    // collect first timer dp
    hipEventRecord(start);
    // run the kernel
    naiveSS<<<1, 1>>>(input, output);
    hipDeviceSynchronize();
    // collect second timer dp
    hipEventRecord(end);

    hipEventSynchronize(end);
    float milsec = 0;
    // calculates total run time in ms
    hipEventElapsedTime(&milsec, start, end);

    printf("elapsed time: %f ms\n", milsec);

    // // check results
    // for (int i = 0; i < SIZE; i++) {
    //     printf("%d ", output[i]);
    // }
    // printf("\n");

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // free mem
    hipFree(input);
    hipFree(output);

    return 0;
}